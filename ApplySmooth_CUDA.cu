#include "hip/hip_runtime.h"
#include "ApplySmooth_CUDA.h"

__global__ void smooth_cuda(unsigned short* cuda_image, unsigned short* new_cuda_image, int rows, int cols){
	int coordX = (blockIdx.x*blockDim.x)+threadIdx.x;
	int coordY = (blockIdx.y*blockDim.y)+threadIdx.y;

	int n = rows*cols;
	int count = 0;
	unsigned int sum = 0;
	for(int i=max(0, coordX-2); i<=min(n-1, coordX+2); i++){
		for(int j=max(0, coordY-2); j<=min(n-1, coordY+2); j++){
			sum += cuda_image[coord(i, j)];
			count++;
		}
	}
	if (count > 0)
		new_cuda_image[coord(coordX, coordY)] = sum/count;
}

void smooth(unsigned short *image, int rows, int cols){
	unsigned short* cuda_image;
	unsigned short* new_cuda_image;

	hipMalloc(&cuda_image, rows*cols*sizeof(unsigned short));
	hipMalloc(&new_cuda_image, rows*cols*sizeof(unsigned short));
	hipMemcpy(cuda_image, image, rows*cols*sizeof(unsigned short), hipMemcpyHostToDevice);
	
	dim3 threadsPerBlock(8,4);
	dim3 numBlocks(rows/threadsPerBlock.x, cols/threadsPerBlock.y);
	smooth_cuda<<<numBlocks, threadsPerBlock>>>(cuda_image, new_cuda_image, rows, cols);

	hipError_t cuda_error = hipGetLastError();
	if (cuda_error != hipSuccess)
		printf("Cuda Error: %s\n", hipGetErrorString(cuda_error));

	hipMemcpy(image, new_cuda_image, rows*cols*sizeof(unsigned short), hipMemcpyDeviceToHost);
	
	hipFree(cuda_image);
	hipFree(new_cuda_image);
	return;
}
